#include <iostream>

#include "./transpose_cpu.h"
#include "./transpose_tiled.cuh"
#include "./transpose_naive.cuh"
#include "./transpose_no_bank_conflicts.cuh"

#include "../common/common.hpp"
#include "../common/Timer.hpp"

#define M         10240U
#define N         1024U
#define TILE_SIZE 32U

float *x_cpu, *y_cpu;
float *x_gpu, *y_gpu, *y_gpu_copyback;

void prepare_input() {
  x_cpu          = new float[M * N];
  y_cpu          = new float[N * M];
  y_gpu_copyback = new float[N * M];
  randomInit(x_cpu, M * N);

  CUDA_CHECK(hipMalloc(&x_gpu, M * N * sizeof(float)));
  CUDA_CHECK(hipMalloc(&y_gpu, N * M * sizeof(float)));

  CUDA_CHECK(hipMemcpy(x_gpu, x_cpu, sizeof(float) * M * N, hipMemcpyHostToDevice));
}

void cleanup() {
  delete[] x_cpu;
  delete[] y_cpu;
  delete[] y_gpu_copyback;
  CUDA_CHECK(hipFree(x_gpu));
  CUDA_CHECK(hipFree(y_gpu));
}

void test_transpose_cpu() {
  Timer::Get()->Start("transpose_cpu");

  transpose_cpu(x_cpu, y_cpu, M, N);

  Timer::Get()->End("transpose_cpu");
}

void test_transpose_naive() {
  randomInit(y_gpu_copyback, M * N);
  CUDA_CHECK(hipDeviceSynchronize());

  Timer::Get()->Start("transpose_naive");

  dim3 block = {TILE_SIZE, TILE_SIZE, 1};
  dim3 grid  = {ceil_div(N, TILE_SIZE), ceil_div(M, TILE_SIZE), 1};
  transpose_naive<<<grid, block>>>(y_gpu, x_gpu, M, N);
  CUDA_CHECK(hipDeviceSynchronize());

  Timer::Get()->End("transpose_naive");

  CUDA_CHECK(hipMemcpy(y_gpu_copyback, y_gpu, sizeof(float) * M * N, hipMemcpyDeviceToHost));

  if (!all_close(y_cpu, y_gpu_copyback, M * N)) {
    fprintf(stderr, "Error: transpose_naive does not match CPU result!\n");
    exit(1);
  }
}

void test_transpose_tiled() {
  randomInit(y_gpu_copyback, M * N);
  CUDA_CHECK(hipDeviceSynchronize());

  Timer::Get()->Start("transpose_tiled");

  dim3 block = {TILE_SIZE, TILE_SIZE, 1};
  dim3 grid  = {ceil_div(N, TILE_SIZE), ceil_div(M, TILE_SIZE), 1};
  transpose_tiled<TILE_SIZE><<<grid, block>>>(y_gpu, x_gpu, M, N);
  CUDA_CHECK(hipDeviceSynchronize());

  Timer::Get()->End("transpose_tiled");

  CUDA_CHECK(hipMemcpy(y_gpu_copyback, y_gpu, sizeof(float) * M * N, hipMemcpyDeviceToHost));

  if (!all_close(y_cpu, y_gpu_copyback, M * N)) {
    fprintf(stderr, "Error: transpose_tiled does not match CPU result!\n");
    // print cpu result
    printf("CPU:\n");
    for (int i = 0; i < N; i++) {
      for (int j = 0; j < M; j++)
        printf("%f ", y_cpu[i * M + j]);
      printf("\n");
    }
    // print gpu result
    printf("GPU:\n");
    for (int i = 0; i < N; i++) {
      for (int j = 0; j < M; j++)
        printf("%f ", y_gpu_copyback[i * M + j]);
      printf("\n");
    }
    exit(1);
  }
}

void test_transpose_no_bc() {
  randomInit(y_gpu_copyback, M * N);
  CUDA_CHECK(hipDeviceSynchronize());

  Timer::Get()->Start("transpose_no_bc");

  dim3 block = {TILE_SIZE, TILE_SIZE, 1};
  dim3 grid  = {ceil_div(N, TILE_SIZE), ceil_div(M, TILE_SIZE), 1};
  transpose_no_bank_conflicts<TILE_SIZE><<<grid, block>>>(y_gpu, x_gpu, M, N);
  CUDA_CHECK(hipDeviceSynchronize());

  Timer::Get()->End("transpose_no_bc");

  CUDA_CHECK(hipMemcpy(y_gpu_copyback, y_gpu, sizeof(float) * M * N, hipMemcpyDeviceToHost));

  if (!all_close(y_cpu, y_gpu_copyback, M * N)) {
    fprintf(stderr, "Error: transpose_no_bc does not match CPU result!\n");
    exit(1);
  }
}

int main() {
  prepare_input();

  test_transpose_cpu();

  test_transpose_naive();
  test_transpose_tiled();
  test_transpose_no_bc();

  cleanup();
  return 0;
}